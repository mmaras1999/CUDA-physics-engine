#include "cuda_utilities.hpp"
#include <exception>
#include <stdexcept>

void cudaCheckError()
{
    hipError_t e = hipGetLastError();

    if (e != hipSuccess) 
    {
        printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e));
        throw std::runtime_error("cuda_error");
    }
}